#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <vector>

#include <cub/device/device_radix_sort.cuh>

#include "common.cuh"

template<typename T>
void debug_store_array(std::filesystem::path filepath,
        std::span<const T> data) {
    auto file = std::ofstream(filepath);
    for (auto i = 0; i < data.size(); ++i) {
        file << data[i] << ',';
    }
    file << ";\n";
}

/// https://graphics.stanford.edu/%7Eseander/bithacks.html#RoundUpPowerOf2
constexpr
auto next_pow2(uint32_t v) {
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    v += v == 0;
    return v;
}

__global__
void initialize_indices(uint *indices, const uint N) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    indices[index] = index;
}

/// Calculate the cell index of each particle.
__global__
void get_cell_index_per_particle(const FloatingPoint *pos_x,
        const FloatingPoint *pos_y, const uint particle_count,
        uint *cell_indices) {
    const auto particle_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (particle_index >= particle_count) {
        return;
    }
    const auto x = pos_x[particle_index];
    const auto y = pos_y[particle_index];
    // Use min() to force particles back inside the grid of cell.
    const auto cell_origin = uint2{
        min(static_cast<uint>(floor(x_to_u(x))), U - 1),
        min(static_cast<uint>(floor(y_to_v(y))), V - 1)
    };
    cell_indices[particle_index] = cell_origin.x + cell_origin.y * U;
}

__global__
void initialize_kernel_data(
        const uint *cell_indices,
        const uint particle_count,
        uint *particle_indices_rel_cell,
        uint *particle_count_per_cell) {
    const auto indirect_particle_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (indirect_particle_index >= particle_count) return;

    __shared__ uint shared_cell_indices[block_size];

    __shared__ uint shared_particle_indices_rel_cell[block_size];
    // Incrementing 0, 1, 2, ..., for each new cell.
    __shared__ uint cell_ids[block_size];
    // Indexed with cell id, not cell index.
    __shared__ uint particle_count_per_cell_id[block_size];

    const auto global_cell_index = cell_indices[indirect_particle_index];
    shared_cell_indices[threadIdx.x] = global_cell_index;
    __syncthreads();

    /* if (blockIdx.x == 16410) {
        printf("----- %d\n", threadIdx.x);
    } */

    if (threadIdx.x == 0) {
        shared_particle_indices_rel_cell[0] = 0;
        cell_ids[0] = 0;
        // Start on 1 since we already set the value for i = 0.
        auto particle_index_rel_cell = 1;
        auto cell_id = 0;
        auto cell_particle_count = 0;
        auto previous_cell_index = shared_cell_indices[0];

        // TODO: Don't loop too far in last block.
        const auto block_particle_count = min(
            particle_count - indirect_particle_index, block_size);
        for (auto i = 1u; i < block_particle_count; ++i) {
            const auto cell_index = shared_cell_indices[i];
            ++cell_particle_count;
            if (cell_index > previous_cell_index) {
                particle_count_per_cell_id[cell_id] = cell_particle_count;
                particle_index_rel_cell = 0;
                ++cell_id;
                cell_particle_count = 0;
                previous_cell_index = cell_index;
            }
            shared_particle_indices_rel_cell[i] = particle_index_rel_cell++;
            cell_ids[i] = cell_id;
        }
        particle_count_per_cell_id[cell_id] = cell_particle_count + 1;
    }
    __syncthreads();
    particle_indices_rel_cell[indirect_particle_index]
        = shared_particle_indices_rel_cell[threadIdx.x];
    const auto cell_index = cell_ids[threadIdx.x];
    particle_count_per_cell[indirect_particle_index]
        = particle_count_per_cell_id[cell_index];
}

__global__
void add_density_shared(
        const FloatingPoint *pos_x,
        const FloatingPoint *pos_y,
        const uint particle_count,
        FloatingPoint *density,
        const uint *cell_indices,
        const uint *particle_indices,
        const uint *particle_indices_rel_cell,
        const uint *particle_count_per_cell
    ) {
    const auto indirect_particle_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (indirect_particle_index >= particle_count) return;

    // Each particle will contribute to 4 cells.
    __shared__ FloatingPoint density_shared[4][block_size];

    const auto cell_index = cell_indices[indirect_particle_index];
    const auto particle_index = particle_indices[indirect_particle_index];
    const auto particle_index_rel_cell = particle_indices_rel_cell[indirect_particle_index];
    const auto cell_particle_count = particle_count_per_cell[indirect_particle_index];
    
    const auto cell_origin = uint2{ cell_index % U, cell_index / U };
    // Node indices.
    const auto indices = uint4{
        get_node_index(cell_origin.x,     cell_origin.y),
        get_node_index(cell_origin.x + 1, cell_origin.y),
        get_node_index(cell_origin.x,     cell_origin.y + 1),
        get_node_index(cell_origin.x + 1, cell_origin.y + 1)
    };

    const auto x = pos_x[particle_index];
    const auto y = pos_y[particle_index];
    const auto u = x_to_u(x);
    const auto v = y_to_v(y);

    // Node weights. https://www.particleincell.com/2010/es-pic-method/
    const auto pos_relative_cell = FloatingPoint2{ u - cell_origin.x, v - cell_origin.y };
    const FloatingPoint weights[] = {
        (1 - pos_relative_cell.x) * (1 - pos_relative_cell.y),
             pos_relative_cell.x  * (1 - pos_relative_cell.y),
        (1 - pos_relative_cell.x) *      pos_relative_cell.y,
             pos_relative_cell.x  *      pos_relative_cell.y
    };

    density_shared[0][threadIdx.x] = weights[0];
    density_shared[1][threadIdx.x] = weights[1];
    density_shared[2][threadIdx.x] = weights[2];
    density_shared[3][threadIdx.x] = weights[3];
    __syncthreads();

    // in-place reduction in shared memory
    // XXX: Assumes both fixed and equal number of particles per cell.
    for (int stride = next_pow2(cell_particle_count) / 2; stride > 0; stride /= 2) {
        // Make sure not to stride outside of the cell range. Crucial when
        // the number of particles in a cell isn't a power of two.
        if (particle_index_rel_cell < stride && particle_index_rel_cell + stride < cell_particle_count) {
            density_shared[0][threadIdx.x] += density_shared[0][threadIdx.x + stride];
            density_shared[1][threadIdx.x] += density_shared[1][threadIdx.x + stride];
            density_shared[2][threadIdx.x] += density_shared[2][threadIdx.x + stride];
            density_shared[3][threadIdx.x] += density_shared[3][threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (particle_index_rel_cell == 0) {
        atomicAdd(&density[indices.x], density_shared[0][threadIdx.x]);
        atomicAdd(&density[indices.y], density_shared[1][threadIdx.x]);
        atomicAdd(&density[indices.z], density_shared[2][threadIdx.x]);
        atomicAdd(&density[indices.w], density_shared[3][threadIdx.x]);
    }
}

int main() {
#ifndef DEBUG_DISTRIBUTION
    // Generate a particle density.
    auto particle_count_per_cell = std::vector<uint>(cell_count);
    const auto N = generate_particle_density(particle_count_per_cell);
#endif
    // Allocate on the host.
    auto h_pos_x = std::vector<FloatingPoint>(N);
    auto h_pos_y = std::vector<FloatingPoint>(N);
    auto h_cell_indices_before = std::vector<uint>(N);
    auto h_cell_indices_after = std::vector<uint>(N);
    auto h_density = std::vector<FloatingPoint>(node_count);

    // Allocate on the device.
    auto d_pos_x = (decltype(h_pos_x)::value_type *){};
    auto d_pos_y = (decltype(h_pos_y)::value_type *){};
    auto d_particle_indices_before = (uint *){};
    auto d_particle_indices_after = (uint *){};
    auto d_cell_indices_before = (decltype(h_cell_indices_before)::value_type *){};
    auto d_cell_indices_after = (decltype(h_cell_indices_before)::value_type *){};
    auto d_particle_indices_rel_cell = (uint *){};
    auto d_particle_count_per_cell = (uint *){};
    auto d_density = (decltype(h_density)::value_type *){};
    allocate_array(&d_pos_x, h_pos_x.size());
    allocate_array(&d_pos_y, h_pos_y.size());
    allocate_array(&d_particle_indices_before, h_pos_x.size());
    allocate_array(&d_particle_indices_after, h_pos_x.size());
    allocate_array(&d_cell_indices_before, h_cell_indices_before.size());
    allocate_array(&d_cell_indices_after, h_cell_indices_before.size());
    allocate_array(&d_particle_indices_rel_cell, h_pos_x.size());
    allocate_array(&d_particle_count_per_cell, h_pos_x.size());
    allocate_array(&d_density, h_density.size());

    const auto block_count = (N + block_size - 1) / block_size;
    printf("N: %d, block_count: %d, block_size: %d\n", N, block_count, block_size);

    // Determine temporary device storage requirements
    void *d_sort_storage = nullptr;
    auto sort_storage_byte_count = size_t{};
    hipcub::DeviceRadixSort::SortPairs(
        d_sort_storage, sort_storage_byte_count,
        d_cell_indices_before, d_cell_indices_after,
        d_particle_indices_before, d_particle_indices_after,
        N
    );
    // Allocate temporary storage
    hipMalloc(&d_sort_storage, sort_storage_byte_count);

    // Distribute the cells using shuffled indices to force uncoalesced global
    // access when reading particle postions.
    const auto distribution_indices = get_shuffled_indices(h_pos_x.size());
    distribute_from_density(h_pos_x, h_pos_y, distribution_indices,
        particle_count_per_cell);

    initialize_indices<<<block_count, block_size>>>(
        d_particle_indices_before, N
    );

    // Copy from the host to the device.
    store(d_pos_x, h_pos_x);
    store(d_pos_y, h_pos_y);

    hipDeviceSynchronize();
    // Perform multiple iterations and pretend the particles are moving as well.
    for (auto i = 0; i < iteration_count; ++i) {
        using namespace std::chrono;
        const auto start_time = high_resolution_clock::now();

        // Reset density.
        fill(d_density, 0, h_density.size());

        get_cell_index_per_particle<<<
            block_count, block_size
        >>>(
            d_pos_x, d_pos_y, N, d_cell_indices_before
        );

        // Run sorting operation
        hipcub::DeviceRadixSort::SortPairs(
            d_sort_storage, sort_storage_byte_count,
            d_cell_indices_before, d_cell_indices_after,
            d_particle_indices_before, d_particle_indices_after,
            N
        );

        initialize_kernel_data<<<
            block_count, block_size
        >>>(
            d_cell_indices_after, N,
            d_particle_indices_rel_cell, d_particle_count_per_cell);

        auto h_particle_indices_rel_cell = std::vector<uint>{};
        auto h_particle_count_per_cell = std::vector<uint>{};
        h_particle_indices_rel_cell.reserve(N);
        h_particle_count_per_cell.reserve(N);
        load(h_particle_indices_rel_cell, d_particle_indices_rel_cell);
        load(h_particle_count_per_cell, d_particle_count_per_cell);

        add_density_shared<<<
            block_count, block_size
        >>>(
            d_pos_x, d_pos_y, N, d_density,
            d_cell_indices_after,
            d_particle_indices_after,
            d_particle_indices_rel_cell,
            d_particle_count_per_cell
        );
        load(h_density, d_density);

        const auto end_time = high_resolution_clock::now();
        const auto duration = end_time - start_time;
        const auto duration_ms = duration_cast<milliseconds>(duration).count();
        const auto duration_us = duration_cast<microseconds>(duration).count();
        if (duration_ms == 0) {
            printf("Iteration %d took %ld us.\n", i,
                duration_us);
        } else {
            printf("Iteration %d took %ld.%ld ms.\n", i,
                duration_ms,
                duration_us);
        }
    }

    // Free device memory.
    hipFree(d_pos_x);
    hipFree(d_pos_y);
    hipFree(d_particle_indices_before);
    hipFree(d_cell_indices_before);
    hipFree(d_density);

    hipFree(d_sort_storage);
    hipFree(d_particle_indices_rel_cell);
    hipFree(d_particle_count_per_cell);

#ifdef DEBUG_STORE_RESULTS
    const auto output_directory = std::filesystem::path("output");
    std::filesystem::create_directory(output_directory);
    store_positions(output_directory / "positions_shared.csv", h_pos_x, h_pos_y);
    store_density(output_directory / "density_shared.csv", h_density);
#endif
}
