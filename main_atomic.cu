#include "hip/hip_runtime.h"
#include <filesystem>
#include <fstream>
#include <iostream>
#include <vector>

#include "common.hpp"

__global__
void add_density_atomic(const FloatingPoint *pos_x, const FloatingPoint *pos_y,
        FloatingPoint *density) {
    const auto index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) {
        return;
    }

    const auto x = pos_x[index];
    const auto y = pos_y[index];
    const auto u = x_to_u(x);
    const auto v = y_to_v(y);

    // Node coordinates.
    const auto node_bottom_left  = int2{ static_cast<int>(floor(u)), static_cast<int>(floor(v)) };
    const auto node_bottom_right = int2{ static_cast<int>( ceil(u)), static_cast<int>(floor(v)) };
    const auto node_top_left     = int2{ static_cast<int>(floor(u)), static_cast<int>( ceil(v)) };
    const auto node_top_right    = int2{ static_cast<int>( ceil(u)), static_cast<int>( ceil(v)) };

    // Node weights. https://www.particleincell.com/2010/es-pic-method/
    const auto pos_relative_cell = FloatingPoint2{ u - node_bottom_left.x, v - node_bottom_left.y };
    const auto weight_bottom_left  = (1 - pos_relative_cell.x) * (1 - pos_relative_cell.y);
    const auto weight_bottom_right =      pos_relative_cell.x  * (1 - pos_relative_cell.y);
    const auto weight_top_left     = (1 - pos_relative_cell.x) *      pos_relative_cell.y;
    const auto weight_top_right    =      pos_relative_cell.x  *      pos_relative_cell.y;

    // Node indices.
    const auto index_bottom_left = get_node_index(node_bottom_left.x, node_bottom_left.y);
    const auto index_bottom_right = get_node_index(node_bottom_right.x, node_bottom_right.y);
    const auto index_top_left = get_node_index(node_top_left.x, node_top_left.y);
    const auto index_top_right = get_node_index(node_top_right.x, node_top_right.y);

    atomicAdd(&density[index_bottom_left], weight_bottom_left);
    atomicAdd(&density[index_bottom_right], weight_bottom_right);
    atomicAdd(&density[index_top_left], weight_top_left);
    atomicAdd(&density[index_top_right], weight_top_right);
}

void store_density(std::filesystem::path filepath,
                   std::span<const FloatingPoint> density) {
    auto density_file = std::ofstream(filepath);
    for (int row = 0; row < (V + 1); ++row) {
        for (int col = 0; col < (U + 1); ++col) {
            density_file << density[row * (U + 1) + col] << ',';
        }
        density_file << '\n';
    }
}

int main() {
    // Allocate particle positions and densities on the host.
    auto h_pos_x = std::vector<FloatingPoint>(positions_count);
    auto h_pos_y = std::vector<FloatingPoint>(positions_count);
    auto h_density = std::vector<FloatingPoint>(lattice_count);

    // Allocate particle positions and densities on the device.
    FloatingPoint *d_pos_x;
    FloatingPoint *d_pos_y;
    FloatingPoint *d_density;
    hipMalloc(&d_pos_x, positions_bytes);
    hipMalloc(&d_pos_y, positions_bytes);
    hipMalloc(&d_density, lattice_bytes);

    distribute_random(h_pos_x, h_pos_y);

    // Copy positions from the host to the device.
    hipMemcpy(d_pos_x, h_pos_x.data(), positions_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_pos_y, h_pos_y.data(), positions_bytes, hipMemcpyHostToDevice);

    // Initialize density.
    hipMemset(d_density, 0, lattice_bytes);

    add_density_atomic<<<block_count, block_size>>>(d_pos_x, d_pos_y, d_density);
    hipMemcpy(h_density.data(), d_density, lattice_bytes, hipMemcpyDeviceToHost);

    // Free device memory.
    hipFree(d_pos_x);
    hipFree(d_pos_y);

    // Store data to files.
    const auto output_directory = std::filesystem::path("output");
    std::filesystem::create_directory(output_directory);
    store_density(output_directory / "density_atomic.csv", h_density);
}
