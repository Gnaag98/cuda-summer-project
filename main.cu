
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>
#include <random>
#include <vector>

int main() {
    // Around 2 million particles.
    const auto N = 1 << 21;

    // Size of 2D space.
    const auto width = 2000.0f;
    const auto height = 4000.0f;
    const auto half_width = width / 2;
    const auto half_height = height / 2;

    const auto random_seed = 1u;

    // Allocation size for 1D arrays.
    const auto size_1d = N * sizeof(float);

    // Allocate particle positions on the host.
    auto h_pos_x = std::vector<float>(size_1d);
    auto h_pos_y = std::vector<float>(size_1d);

    // Allocate particle positions on the device.
    float *d_pos_x;
    float *d_pos_y;
    hipMalloc(&d_pos_x, size_1d);
    hipMalloc(&d_pos_y, size_1d);

    // Randomly distribute particles in 2D space.
    auto random_engine = std::default_random_engine(random_seed);
    auto uniform_distribution_x = std::uniform_real_distribution<float>(-half_width, half_width);
    auto uniform_distribution_y = std::uniform_real_distribution<float>(-half_height, half_height);
    for (size_t i = 0; i < size_1d; ++i) {
        h_pos_x[i] = uniform_distribution_x(random_engine);
        h_pos_y[i] = uniform_distribution_y(random_engine);
    }

    for (int i = 0; i < 10; ++i) {
        std::cout << std::setw(10) << h_pos_x[i]
                  << std::setw(10) << h_pos_y[i]
                  << '\n';
    }

    // Free device memory.
    hipFree(d_pos_x);
    hipFree(d_pos_y);
}
